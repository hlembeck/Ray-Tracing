#include "hip/hip_runtime.h"
#include "LightColor.cuh"

//x: wavelength (nm)
double redSpectrum(double x) {
	return .25 * exp(.5 * (x - 620) * (x - 620) / 900.0);
}

double greenSpectrum(double x) {
	return .2 * exp(.5 * (x - 535.0) * (x - 535.0) / 900.0);
}

double blueSpectrum(double x) {
	return .45 * exp(-.5 * (x - 460.0) * (x - 460.0) / 900.0);
}

double colorMatch_X(double l) {
	return 1.056 * piecewiseGaussian(l, 599.8, 37.9, 31.0) + .362 * piecewiseGaussian(l, 442.0, 16.0, 26.7) - .065 * piecewiseGaussian(l, 501.1, 20.4, 26.2);
}

double colorMatch_Y(double l) {
	return 0.821 * piecewiseGaussian(l, 568.8, 46.9, 40.5) + .286 * piecewiseGaussian(l, 530.9, 16.3, 31.1);
}

double colorMatch_Z(double l) {
	return 1.217 * piecewiseGaussian(l, 437.0, 11.8, 36.0) + .681 * piecewiseGaussian(l, 459.0, 26.0, 13.8);
}

RGBa* XYZ::toRGB() {
	double* mat = new double[9];

	mat[0] = fg_RiemannSum(MIN_WAVELENGTH, MAX_WAVELENGTH, 1, &redSpectrum, &colorMatch_X);
	mat[1] = fg_RiemannSum(MIN_WAVELENGTH, MAX_WAVELENGTH, 1, &redSpectrum, &colorMatch_Y);
	mat[2] = fg_RiemannSum(MIN_WAVELENGTH, MAX_WAVELENGTH, 1, &redSpectrum, &colorMatch_Z);

	mat[3] = fg_RiemannSum(MIN_WAVELENGTH, MAX_WAVELENGTH, 1, &greenSpectrum, &colorMatch_X);
	mat[4] = fg_RiemannSum(MIN_WAVELENGTH, MAX_WAVELENGTH, 1, &greenSpectrum, &colorMatch_Y);
	mat[5] = fg_RiemannSum(MIN_WAVELENGTH, MAX_WAVELENGTH, 1, &greenSpectrum, &colorMatch_Z);

	mat[6] = fg_RiemannSum(MIN_WAVELENGTH, MAX_WAVELENGTH, 1, &blueSpectrum, &colorMatch_X);
	mat[7] = fg_RiemannSum(MIN_WAVELENGTH, MAX_WAVELENGTH, 1, &blueSpectrum, &colorMatch_Y);
	mat[8] = fg_RiemannSum(MIN_WAVELENGTH, MAX_WAVELENGTH, 1, &blueSpectrum, &colorMatch_Z);

	Matrix m(mat);
	Vector rgb = m.left_transform(Vector(_X, _Y, _Z));
	return new RGBa(rgb.x, rgb.y, rgb.z, 255);
}

XYZ PiecewiseSpectrum::toXYZ() {
	double* tVals = new double[STEPS];
	double x, y, z;
	for (unsigned short i = 0; i < STEPS; i++) {
		tVals[i] = _power[i] * colorMatch_X(MIN_WAVELENGTH + STEP_LENGTH * i);
	}
	x = rSum(MIN_WAVELENGTH, MAX_WAVELENGTH, STEP_LENGTH, tVals);

	for (unsigned short i = 0; i < STEPS; i++) {
		tVals[i] = _power[i] * colorMatch_Y(MIN_WAVELENGTH + STEP_LENGTH * i);
	}
	y = rSum(MIN_WAVELENGTH, MAX_WAVELENGTH, STEP_LENGTH, tVals);

	for (unsigned short i = 0; i < STEPS; i++) {
		tVals[i] = _power[i] * colorMatch_Z(MIN_WAVELENGTH + STEP_LENGTH * i);
	}
	z = rSum(MIN_WAVELENGTH, MAX_WAVELENGTH, STEP_LENGTH, tVals);

	return XYZ(x, y, z);
}